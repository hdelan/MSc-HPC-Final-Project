#include "hip/hip_runtime.h"
/**
 * \file:        cu_lanczos.cu
 * \brief:       Perform Lanczos decomposition in parallel. Complete with optimal blocksizes for each kernel
 * \author:      Hugh Delaney
 * \version:     
 * \date:        2021-09-16
 */

#include "cu_lanczos.h"

// Optimal blocksizes
#define SPMV_BLOCKSIZE 1024
#define DOT_BLOCKSIZE 128
#define NORM_BLOCKSIZE 128
#define RED_BLOCKSIZE 256
#define SAX_BLOCKSIZE 256


  template <typename T>
void lanczosDecomp<T>::cu_decompose()
{
  unsigned n{A.get_n()};
  unsigned *IA_d, *JA_d;
  T *v_d, *alpha_d, *beta_d, *tmp_d;

  unsigned dot_grid {n/(DOT_BLOCKSIZE*2) + (n%(DOT_BLOCKSIZE*2)==0?0:1)};
  unsigned norm_grid {n/(NORM_BLOCKSIZE*2) + (n%(NORM_BLOCKSIZE*2)==0?0:1)};
  unsigned sax_grid {n/SAX_BLOCKSIZE + (n%SAX_BLOCKSIZE==0?0:1)};

  T *x_normed{new T[n]};
  T x_norm = norm(x, n);

  for (auto k = 0u; k < n; k++)
    x_normed[k] = x[k] / x_norm;
  
  hipError_t cudaStat;
  cudaStat = hipMalloc((void **)&Q_d, sizeof(T) * n * 2);
  if (cudaStat != hipSuccess) {
    std::cerr << "Could not allocate space for Q_d!\n";
    return;
  }

  cudaStat = hipMalloc((void **)&IA_d, sizeof(unsigned) * (n + 1));
  if (cudaStat != hipSuccess) {
    std::cerr << "Could not allocate space for IA_d!\n";
    return;
  }
  cudaStat = hipMalloc((void **)&JA_d, sizeof(unsigned) * 2 * A.edge_count);
  if (cudaStat != hipSuccess) {
    std::cerr << "Could not allocate space for JA_d!\n";
    return;
  }
  cudaStat = hipMalloc((void **)&v_d, sizeof(T) * n);
  if (cudaStat != hipSuccess) {
    std::cerr << "Could not allocate space for v_d!\n";
    return;
  }
  cudaStat = hipMalloc((void **)&alpha_d, sizeof(T) * krylov_dim);
  if (cudaStat != hipSuccess) {
    std::cerr << "Could not allocate space for alpha_d!\n";
    return;
  }
  cudaStat = hipMalloc((void **)&beta_d, sizeof(T) * (krylov_dim - 1));
  if (cudaStat != hipSuccess) {
    std::cerr << "Could not allocate space for beta_d!\n";
    return;
  }
  cudaStat = hipMalloc((void **)&tmp_d, sizeof(T) * (norm_grid));
  if (cudaStat != hipSuccess) {
    std::cerr << "Could not allocate space for tmp_d!\n";
    return;
  }

  auto global_memory_used { (n+1+2*A.edge_count)*sizeof(unsigned)+(n+2*krylov_dim-1+norm_grid)*sizeof(T)};

  std::cout << "\nUsing " 
        << global_memory_used << " bytes of CUDA global memory (" 
        << 100* global_memory_used/(double)11996954624 
        << "% of capacity 11996954624 bytes)\n";

  T *Q_d_ptr[2] = {&Q_d[0], &Q_d[n]};
  
  hipStream_t stream[3];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);

  hipMemcpyAsync(Q_d_ptr[0], x_normed, sizeof(T) * n, hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(IA_d, A.row_offset, sizeof(unsigned) * (n + 1), hipMemcpyHostToDevice, stream[1]);
  hipMemcpyAsync(JA_d, A.col_idx, sizeof(unsigned) * 2 * A.edge_count, hipMemcpyHostToDevice, stream[2]);
  
  hipStreamSynchronize(stream[0]);
  hipStreamSynchronize(stream[1]);
  hipStreamSynchronize(stream[2]);
  
  int i {0};
  for (auto k = 0u; k < krylov_dim; k++)
  {
    // v = A*Q(:,j)
    // SPMV1 is used instead of SPMV2 since it uses less memory
    cu_spMV1<T><<<n/256+1,256>>>(IA_d, JA_d, n, Q_d_ptr[i], v_d);

    // alpha = v*Q(:,j)
    cu_dot_prod<T, DOT_BLOCKSIZE><<<dot_grid, DOT_BLOCKSIZE,0,stream[0]>>>(v_d, Q_d_ptr[i], n, tmp_d);
    cu_reduce<T, RED_BLOCKSIZE><<<1, RED_BLOCKSIZE, 0, stream[0]>>>(tmp_d, dot_grid, &alpha_d[k]);

    // v = v - alpha*Q(:,j)
    cu_dpax<T><<<sax_grid, SAX_BLOCKSIZE,0,stream[0]>>>(v_d, &alpha_d[k], Q_d_ptr[i], n);

    if (k > 0)
    {
      // v = v - beta*Q(:,j-1)
      cu_dpax<T><<<sax_grid, SAX_BLOCKSIZE,0,stream[0]>>>(v_d, &beta_d[k - 1], Q_d_ptr[1-i], n);
    }

    if (k < krylov_dim - 1)
    {
      // beta[j] = norm(v)
      cu_norm_sq<T, NORM_BLOCKSIZE><<<norm_grid, NORM_BLOCKSIZE, 0, stream[0]>>>(v_d, n, tmp_d);
      cu_reduce_sqrt<T,RED_BLOCKSIZE><<<1, RED_BLOCKSIZE, 0, stream[0]>>>(tmp_d, norm_grid, &beta_d[k]);
      
      // Q(:,j) = v/beta
      cu_dvexda<T><<<sax_grid,SAX_BLOCKSIZE,0,stream[0]>>>(Q_d_ptr[1-i], &beta_d[k], v_d, n);
    }
    // Send back columns of Q every cycle
    hipMemcpyAsync(&Q[k*n], Q_d_ptr[i], sizeof(T) * n, hipMemcpyDeviceToHost, stream[2]);
    i = 1 - i;
  }
  hipMemcpyAsync(alpha, alpha_d, sizeof(T) * krylov_dim, hipMemcpyDeviceToHost, stream[0]);
  hipMemcpyAsync(beta, beta_d, sizeof(T) * (krylov_dim - 1), hipMemcpyDeviceToHost, stream[1]);

  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  hipStreamDestroy(stream[2]);
  
  hipFree(IA_d);
  hipFree(JA_d);
  hipFree(v_d);
  hipFree(alpha_d);
  hipFree(beta_d);
  hipFree(tmp_d);
  hipFree(blockrows_d);
}

template void lanczosDecomp<float>::cu_decompose();
template void lanczosDecomp<double>::cu_decompose();

