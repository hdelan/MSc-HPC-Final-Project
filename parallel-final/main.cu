/**
 * \file:        main.cu
 * \brief:       The final implementation of CUDA Lanczos method.
 * \author:      Hugh Delaney
 * \version:     
 * \date:        2021-09-16
 */
#include "lib/cu_linalg.h"
#include "lib/cu_lanczos.h"
#include "lib/cu_SPMV.h"
#include "lib/multiplyOut.h"
#include "lib/eigen.h"
#include "lib/SPMV.h"
#include "lib/adjMatrix.h"
#include "lib/check_ans.h"
#include "lib/helpers.h"
#include "lib/write_ans.h"

#include <iostream>
#include <fstream>
#include <iomanip>
#include <random>
#include <vector>
#include <algorithm>
#include <sys/time.h>
#include <string>

#include <hip/hip_runtime_api.h>

#define WIDTH 81  // for formatting std::cout output

double seconds_between(timeval s, timeval e);

int main(int argc, char ** argv)
{
    unsigned n{10'000};
    unsigned deg{5};
    unsigned edges{n * 10};
    unsigned krylov_dim {100};
    unsigned width {17};
    bool verbose {true};
    adjMatrix A;
    
    timeval start, end;
    gettimeofday(&start, NULL);
    
    //std::string filename {"California"};
    std::string filename {"bn1000000e9999944"};
    //std::string filename {"europe_osm"};
    //std::string filename {"delaunay_n24"};

    parseArguments(argc, argv, filename, krylov_dim, verbose, n, deg, edges);

    std::string filepath = "../data/"+filename+"/"+filename+".mtx";
    std::cout << "Going to open file: " << filepath << std::endl;

    char make_or_read_matrix {'f'};
    if (make_or_read_matrix == 'f') {
      std::ifstream fs;
      fs.open(filepath);
      assert(!fs.fail() && "File opening failed\n");
      fs >> n >> n >> edges;
      adjMatrix B(n, edges, fs);
      A = std::move(B);
      fs.close();
    } else {
      unsigned barabasi_degree {20};
      // Make random matrix
      adjMatrix B(n, barabasi_degree, 'b');
      A = std::move(B);
    }

    gettimeofday(&end, NULL);
    std::cout << "\nTime elapsed to build random adjacency matrix with n = " << n << " edges = " << edges << ":\n\t"
      << seconds_between(start, end) << " seconds\n\n";

    std::cout << "Running Lanczos algorithm for krylov_dim "<< krylov_dim << "\n\n";

    std::vector<double> x_double (n, 1);
    std::vector<float> x_float (n, 1);
    
    timeval s, e1, e2, e3, e4;
    gettimeofday(&s, NULL);
    
    // SERIAL LANCZOS
    bool cuda = false;
    lanczosDecomp<double> L(A, krylov_dim, &x_double[0], cuda);
    gettimeofday(&e1, NULL);

    eigenDecomp<double> E(L);
    gettimeofday(&e2, NULL);

    multOut(L, E, A, false);
    gettimeofday(&e3, NULL);

    gettimeofday(&e4, NULL);

    double cpu_time_lanczos {seconds_between(s, e1)};
    double cpu_time_mult {seconds_between(e2, e3)};
    double cpu_time_whole {seconds_between(s, e3)};
    
    timeval s_d, e_d;
    hipEvent_t start1_d, start2_d, end1_d, end2_d;
    cuda_start_timer(start1_d, end1_d);
    
    L.free_mem();

    // CUDA LANCZOS
    cuda = true;
    
    // To use floats uncomment the following line
    //lanczosDecomp<float> cu_L(A, krylov_dim, &x_float[0], cuda);
    
    // And comment out the below
    lanczosDecomp<double> cu_L(A, krylov_dim, &x_double[0], cuda);
    float gpu_time_lanczos{cuda_end_timer(start1_d, end1_d)};

    gettimeofday(&s_d, NULL);
    
    // To use floats uncomment the following line
    //eigenDecomp<float> cu_E(cu_L);
    
    // And comment out the below
    eigenDecomp<double> cu_E(cu_L);

    cuda_start_timer(start2_d, end2_d);
    multOut(cu_L, cu_E, A, true);
    float gpu_time_mult{cuda_end_timer(start2_d, end2_d)};

    gettimeofday(&e_d, NULL);
    double gpu_time_whole {gpu_time_lanczos + (e_d.tv_sec - s_d.tv_sec + (e_d.tv_usec - s_d.tv_usec) / 1000000.0)};
    

    std::cout << std::setfill('~') << std::setw(WIDTH) << '\n' << std::setfill(' ');
    std::cout << "TIMING\n";
    std::cout << std::setfill('~') << std::setw(WIDTH) << '\n' << std::setfill(' ');
    std::cout << std::setw(2*width) << "Serial" << std::setw(width) << "CUDA" << std::setw(width) << "Speedup"<<'\n';
    std::cout << std::setfill('~') << std::setw(WIDTH) << '\n' << std::setfill(' ');
    std::cout << std::setw(width) << std::left << "Lanczos" << std::right
      << std::setw(width) << cpu_time_lanczos 
      << std::setw(width) << gpu_time_lanczos
      << std::setw(width) << cpu_time_lanczos/gpu_time_lanczos << "\n\n";
    std::cout << std::setw(width) << std::left << "Multiply Out" << std::right
      << std::setw(width) << cpu_time_mult
      << std::setw(width) << gpu_time_mult
      << std::setw(width) << cpu_time_mult/gpu_time_mult << "\n\n";
    std::cout << std::setw(width) << std::left << "Entire algorithm" << std::right
      << std::setw(width) << cpu_time_whole
      << std::setw(width) << gpu_time_whole
      << std::setw(width) << cpu_time_whole/gpu_time_whole << "\n\n";

    std::cout << std::setfill('~') << std::setw(WIDTH) << '\n' << std::setfill(' ');
    std::cout << "ERROR CHECKING\n";
    std::cout << std::setfill('~') << std::setw(WIDTH) << '\n' << std::setfill(' ');

    check_ans(L, cu_L);
    
    std::string ans_filename = "../data/"+filename+"/ans"+std::to_string(krylov_dim)+".txt";
    write_ans(ans_filename, L);

    std::cout << std::setfill('~') << std::setw(WIDTH) << '\n' << std::setfill(' ');
}

double seconds_between(timeval s, timeval e) {
  return e.tv_sec - s.tv_sec + (e.tv_usec - s.tv_usec) / 1000000.0;
}
