#include "hip/hip_runtime.h"
#include "cu_lanczos.h"
#include "blocks.h"
__global__ void change_IA_for_device1(unsigned * IA_d, const unsigned n) {
  auto tid {blockIdx.x*blockDim.x+threadIdx.x};
  if (tid < n) {
    auto offset {IA_d[0]};
    IA_d[tid] -= offset;
  }
}

/*
__global__ void print_n(float*a, const unsigned n, const int dev) {
  printf("\n");
  printf("Device %d\t", dev);
  for (auto i=0u;i<n;i++)
    printf(" %E ", a[i]);
  printf("\n");
}
__global__ void print_n(double*a, const unsigned n, const int dev) {
  printf("\n");
  printf("Device %d\t", dev);
  for (auto i=0u;i<n;i++)
    printf(" %E ", a[i]);
  printf("\n");
}
__global__ void print_n(unsigned *a, const unsigned n, const int dev) {
  printf("\n");
  printf("Device %d\t", dev);
  for (auto i=0u;i<n;i++)
    printf(" %u ", a[i]);
  printf("\n");
}
*/
  template <typename T>
void lanczosDecomp<T>::cu_decompose()
{
  unsigned n{A.get_n()};
  unsigned *IA_d0, *JA_d0;
  T *v_d0, *alpha_d0, *beta_d0, *tmp_d0, *Q_d_raw0;
  unsigned *IA_d1, *JA_d1;
  T *v_d1, *x_d1;

  T *x_normed{new T[n]};
  T x_norm = norm(x, n);

  for (auto k = 0u; k < n; k++)
    x_normed[k] = x[k] / x_norm;

  std::cout << "Running with "<< BLOCKSIZE << " threads per block\n";

  hipStream_t stream[2];
  hipStream_t memcpy_stream;

  int count;
  hipGetDeviceCount(&count);
  std::cout << "Launching lanczos algorithm on " << count << " cards.\n";

  auto load_balance {0.5}; // This will determine the split of work between card one and card two
  // Card one will receive rows 0 to rows0-1, and card two will receive rows rows0 to n
  unsigned rows0 {static_cast<unsigned> (load_balance*n)};
  unsigned rows1 {n-rows0};
  unsigned edges0 {A.row_offset[rows0]};
  unsigned edges1 {2*A.edge_count - A.row_offset[rows0]};

  assert(edges0+edges1 == 2*A.edge_count);

  int i {0};

  hipSetDevice(0);
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&memcpy_stream);

  unsigned num_blocks_total{static_cast<unsigned>(n) / BLOCKSIZE + 1};
  unsigned num_blocks0{static_cast<unsigned>(rows0) / BLOCKSIZE + 1};

  hipMalloc((void **)&Q_d_raw0, sizeof(T)*n*2);

  hipMalloc((void **)&IA_d0, sizeof(unsigned) * (rows0 + 1));
  hipMalloc((void **)&JA_d0, sizeof(unsigned) * edges0);
  hipMalloc((void **)&v_d0, sizeof(T) * n);
  hipMalloc((void **)&alpha_d0, sizeof(T) * krylov_dim);
  hipMalloc((void **)&beta_d0, sizeof(T) * (krylov_dim - 1));
  hipMalloc((void **)&tmp_d0, sizeof(T) * (num_blocks_total));

  T *Q_d_ptr0[2] {&Q_d_raw0[0], &Q_d_raw0[n]};

  std::vector<T> tmp(n);

  hipMemcpyAsync(Q_d_ptr0[0], x_normed, sizeof(T) * n, hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(IA_d0, A.row_offset, sizeof(unsigned) * (rows0 + 1), hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(JA_d0, A.col_idx, sizeof(unsigned) * edges0, hipMemcpyHostToDevice, stream[0]);

  hipSetDevice(1);
  hipStreamCreate(&stream[1]);

  unsigned num_blocks1 {static_cast<unsigned>(rows1) / BLOCKSIZE + 1};

  hipMalloc((void **)&IA_d1, sizeof(unsigned) * (rows1 + 1));
  hipMalloc((void **)&JA_d1, sizeof(unsigned) * edges1);
  hipMalloc((void **)&v_d1, sizeof(T) * n);
  hipMalloc((void **)&x_d1, sizeof(T) * n);

  hipMemcpyAsync(x_d1, x_normed, sizeof(T) * n, hipMemcpyHostToDevice, stream[1]);
  hipMemcpyAsync(IA_d1, &A.row_offset[rows0], sizeof(unsigned) * (rows1 + 1), hipMemcpyHostToDevice, stream[1]);
  hipMemcpyAsync(JA_d1, &A.col_idx[edges0], sizeof(unsigned) * edges1, hipMemcpyHostToDevice, stream[1]);

  change_IA_for_device1<<<num_blocks1, BLOCKSIZE, 0, stream[1]>>>(IA_d1, rows1+1);
  //print_n<<<1,1,0,stream[1]>>>(&IA_d1[rows1], 1);

  for (auto k = 0u; k < krylov_dim; k++)
  {
    hipSetDevice(1);
    hipStreamSynchronize(stream[0]);
    // v = A*Q(:,j)
    hipSetDevice(0);
    cu_spMV1<T, unsigned><<<num_blocks0, BLOCKSIZE, 0, stream[0]>>>(IA_d0, JA_d0, rows0, Q_d_ptr0[i], v_d0); 
    hipSetDevice(1);
    cu_spMV1<T, unsigned><<<num_blocks1, BLOCKSIZE, 0, stream[1]>>>(IA_d1, JA_d1, rows1, x_d1, v_d1); 
    
    //if (k<10) print_n<<<1,1,0,stream[1]>>>(&v_d1[0], 2);

    hipSetDevice(0);
    hipMemcpyPeer(&v_d0[rows0], 0, v_d1, 1, sizeof(T)*rows1);
    
    //if (k<10) print_n<<<1,1,0,stream[0]>>>(&v_d0[0], 5);
    //if (k<10) print_n<<<1,1,0,stream[0]>>>(Q_d_ptr0[0], 5, 0);

    // alpha = v*Q(:,j)
    if (num_blocks_total==1) { 
      cu_dot_prod<T, BLOCKSIZE><<<1, BLOCKSIZE, BLOCKSIZE*sizeof(T), stream[0]>>>(v_d0, Q_d_ptr0[i], n, &alpha_d0[k]);
    } else {
      cu_dot_prod<T, BLOCKSIZE><<<num_blocks_total/2, BLOCKSIZE, BLOCKSIZE*sizeof(T), stream[0]>>>(v_d0, Q_d_ptr0[i], n, tmp_d0);
      cu_reduce<T, BLOCKSIZE><<<1, BLOCKSIZE, BLOCKSIZE*sizeof(T), stream[0]>>>(tmp_d0, num_blocks_total/2, &alpha_d0[k]);
    }

    //if (k < 10)print_n<<<1,1>>>(&alpha_d0[k], 1, 0);

    // v = v - alpha*Q(:,j)
    cu_dpax<T><<<num_blocks_total, BLOCKSIZE,0,stream[0]>>>(v_d0, &alpha_d0[k], Q_d_ptr0[i], n);

    if (k > 0)
    {
      // v = v - beta*Q(:,j-1)
      cu_dpax<T><<<num_blocks_total, BLOCKSIZE,0,stream[0]>>>(v_d0, &beta_d0[k-1], Q_d_ptr0[1-i], n);
    }

    if (k < krylov_dim - 1)
    {
      // beta[j] = norm(v)
      if (num_blocks_total==1) {
        cu_norm_sq_sqrt<T, BLOCKSIZE><<<1, BLOCKSIZE, BLOCKSIZE*sizeof(T), stream[0]>>>(v_d0, n, &beta_d0[k]);
      } else {
        cu_norm_sq<T, BLOCKSIZE><<<num_blocks_total/2, BLOCKSIZE, BLOCKSIZE*sizeof(T), stream[0]>>>(v_d0, n, tmp_d0);
        cu_reduce_sqrt<T,BLOCKSIZE><<<1, BLOCKSIZE, BLOCKSIZE*sizeof(T), stream[0]>>>(tmp_d0, num_blocks_total, &beta_d0[k]);
      }
      //if (k < 10)print_n<<<1,1>>>(&beta_d0[k], 1,0);

      // Q(:,j) = v/beta
      cu_dvexda<T><<<num_blocks_total,BLOCKSIZE,0,stream[0]>>>(Q_d_ptr0[1-i], &beta_d0[k], v_d0, n);
      //print_n<<<1,1,0,stream[0]>>>(Q_d_ptr0[1-i], 5,0);
      hipSetDevice(1);
      hipMemcpyPeer(x_d1,1, Q_d_ptr0[1-i], 0, sizeof(T)*n);
      //print_n<<<1,1,0,stream[1]>>>(x_d1, 5,1);
      
    }

    hipSetDevice(0);
    hipMemcpy(&tmp[0], Q_d_ptr0[i], sizeof(T)*n, hipMemcpyDeviceToHost);
    i = 1-i;

    for (auto j=0u;j<n;j++)
      Q[k+j*krylov_dim] = tmp[j];
  }
  hipMemcpyAsync(alpha, alpha_d0, sizeof(T) * krylov_dim, hipMemcpyDeviceToHost, stream[0]);
  hipMemcpyAsync(beta, beta_d0, sizeof(T) * (krylov_dim - 1), hipMemcpyDeviceToHost, stream[0]);

  /*
     std::cout << "cu_Q:\n";
     for (int i=0;i<krylov_dim;i++) {
     for (int j=0;j<krylov_dim;j++)
     std::cout << Q[i*krylov_dim+j] << " ";
     std::cout << '\n';
     }

     std::cout << "\ncu_Alpha:\n";
     for (int i=0;i<krylov_dim;i++) std::cout << alpha[i] << " ";

     std::cout << "\n\ncu_Beta:\n";
     for (int i=0;i<krylov_dim-1;i++) std::cout << beta[i] << " ";
     std::cout << "\n\n";

   */
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  hipStreamDestroy(memcpy_stream);

  hipSetDevice(0);
  hipFree(IA_d0);
  hipFree(JA_d0);
  hipFree(v_d0);
  hipFree(Q_d_raw0);
  hipFree(alpha_d0);
  hipFree(beta_d0);
  hipFree(tmp_d0);

  hipSetDevice(1);
  hipFree(IA_d1);
  hipFree(JA_d1);
  hipFree(v_d1);
  hipFree(x_d1);
}

template void lanczosDecomp<float>::cu_decompose();
template void lanczosDecomp<double>::cu_decompose();

 
 
