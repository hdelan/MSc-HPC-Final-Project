#include "hip/hip_runtime.h"
#include "../lib/cu_linalg.h"
#include "../lib/cu_SPMV.h"
#include "../lib/SPMV.h"
#include "../lib/adjMatrix.h"
#include "../lib/helpers.h"

#include <iostream>
#include <iomanip>
#include <random>
#include <vector>
#include <algorithm>
#include <sys/time.h>

#include <hip/hip_runtime_api.h>

#define BLOCKSIZE 4
#define SEED 1234 // To seed RNG
#define WIDTH 81  // for formatting std::cout output

template <typename T>
void cu_linalg_test(const unsigned n, adjMatrix &A);

void cuda_start_timer(hipEvent_t &start, hipEvent_t &end);
float cuda_end_timer(hipEvent_t &start, hipEvent_t &end);

int main(void)
{
    unsigned n{10'000};

    long unsigned edges{n * 100};
    timeval start, end;
    gettimeofday(&start, NULL);
    adjMatrix A(n, edges);
    gettimeofday(&end, NULL);
    std::cout << "Time elapsed for build random adjacency matrix with n = " << n << " edges = " << edges << ": "
              << end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0 << " seconds\n\n";

    std::cout << "\nTesting CUDA vs serial execution of linalg functions for n = " << n << "\n\n";
    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n'
              << std::setfill(' ');
    std::cout << "SINGLE PRECISION\n";
    cu_linalg_test<float>(n, A);
    std::cout << "\n";
    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n'
              << std::setfill(' ');
    std::cout << "DOUBLE PRECISION\n";
    cu_linalg_test<double>(n, A);
}

template <typename T>
void cu_linalg_test(const unsigned n, adjMatrix &A)
{
    T ans;

    std::random_device rd;
    std::mt19937 gen{SEED};
    //std::mt19937 gen{rd()};
    std::uniform_real_distribution<T> U(0.0, 1.0);

    std::vector<T> x(n), y(n), ans_vec(n);

    for (auto it = x.begin(); it != x.end(); it++)
        *it = U(gen);

    for (auto it = y.begin(); it != y.end(); it++)
        *it = U(gen);

    unsigned block_size{BLOCKSIZE}, num_blocks{n / block_size + (n % block_size ? 1 : 0)}, h_blocks{n / (2 * block_size) + (n % (2 * block_size) ? 1 : 0)};

    dim3 blocks{num_blocks}, half_blocks{h_blocks}, threads{block_size}, one_block{1u};

    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n'
              << std::setfill(' ');
    std::cout << "\t\tCUDA\t\tSerial\t\tRelative Error\t\tSpeedup\n";
    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n'
              << std::setfill(' ');

    T *x_d, *y_d, *tmp_d, *ans_d, *spMV_ans_d;
    long unsigned *IA_d, *JA_d;

    hipMalloc((void **)&x_d, sizeof(T) * n);
    hipMalloc((void **)&y_d, sizeof(T) * n);
    hipMalloc((void **)&tmp_d, sizeof(T) * num_blocks);
    hipMalloc((void **)&ans_d, sizeof(T));
    hipMemcpy(x_d, &x[0], sizeof(T) * n, hipMemcpyHostToDevice);
    hipMemcpy(y_d, &y[0], sizeof(T) * n, hipMemcpyHostToDevice);
    {
        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_dot_prod<T, BLOCKSIZE><<<half_blocks, threads, block_size * sizeof(T)>>>(x_d, y_d, n, tmp_d);
        cu_reduce<T, BLOCKSIZE><<<1, threads, block_size * sizeof(T)>>>(tmp_d, num_blocks, ans_d);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);

        timeval start, end;
        gettimeofday(&start, NULL);
        auto serial_ans = std::inner_product(x.begin(), x.end(), y.begin(), 0.0);
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        std::cout << "Inner product: \t" << ans
                  << "  \t" << serial_ans << "\t\t"
                  << (serial_ans - ans) / serial_ans << "\t\t" << speedup << "\n\n";
    }
    {
        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_norm_sq<T, BLOCKSIZE><<<half_blocks, threads, block_size * sizeof(T)>>>(x_d, n, tmp_d);
        cu_reduce<T, BLOCKSIZE><<<1, threads, block_size * sizeof(T)>>>(tmp_d, num_blocks, ans_d);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);

        timeval start, end;
        gettimeofday(&start, NULL);
        auto serial_ans = std::inner_product(x.begin(), x.end(), x.begin(), 0.0);
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        std::cout << "Norm squared: \t" << ans
                  << "  \t" << serial_ans << "\t\t"
                  << (serial_ans - ans) / serial_ans << "\t\t" << speedup << "\n\n";
    }
    {
        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_reduce<T, BLOCKSIZE><<<half_blocks, threads, block_size * sizeof(T)>>>(x_d, n, tmp_d);
        cu_reduce<T, BLOCKSIZE><<<1, threads, block_size * sizeof(T)>>>(tmp_d, num_blocks, ans_d);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);

        timeval start, end;
        gettimeofday(&start, NULL);
        auto serial_ans = std::accumulate(x.begin(), x.end(), 0.0);
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        std::cout << "Reduce: \t" << ans
                  << "  \t" << serial_ans << "\t\t"
                  << (serial_ans - ans) / serial_ans << "\t\t" << speedup << "\n\n";
    }
    {
        hipMemcpy(x_d, &x[0], sizeof(T) * n, hipMemcpyHostToDevice);
        hipMemcpy(y_d, &y[0], sizeof(T) * n, hipMemcpyHostToDevice);

        T alpha{2};
        std::vector<T> gpu_ans_vec(n);

        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_dpax<T><<<blocks, threads>>>(x_d, alpha, y_d, n);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&gpu_ans_vec[0], x_d, n * sizeof(T), hipMemcpyDeviceToHost);

        auto ans{std::inner_product(gpu_ans_vec.begin(), gpu_ans_vec.end(), gpu_ans_vec.begin(), 0.0)};


        int idx{0};
        timeval start, end;
        gettimeofday(&start, NULL);
        std::for_each(x.begin(), x.end(), [&](T & a)
                      { a -= alpha * y[idx++]; });
        auto serial_ans{std::inner_product(x.begin(), x.end(), x.begin(), 0.0)};
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        std::cout << "Vector add: \t" << ans
                  << "  \t" << serial_ans << "\t\t"
                  << (serial_ans - ans) / serial_ans << "\t\t\t" << speedup << "\n\n";
    }
    {
        hipMemcpy(x_d, &x[0], sizeof(T) * n, hipMemcpyHostToDevice);
        hipMemcpy(y_d, &y[0], sizeof(T) * n, hipMemcpyHostToDevice);

        T alpha{4};
        std::vector<T> gpu_ans_vec(n);

        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_dvexda<T><<<blocks, threads>>>(x_d, alpha, y_d, n);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&gpu_ans_vec[0], x_d, n * sizeof(T), hipMemcpyDeviceToHost);

        auto ans{std::inner_product(gpu_ans_vec.begin(), gpu_ans_vec.end(), gpu_ans_vec.begin(), 0.0)};


        int idx{0};
        timeval start, end;
        gettimeofday(&start, NULL);
        std::for_each(x.begin(), x.end(), [&](T & a)
                      { a = y[idx++]/alpha; });
        auto serial_ans{std::inner_product(x.begin(), x.end(), x.begin(), 0.0)};
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        std::cout << "v=v/scalar: \t" << ans
                  << "  \t" << serial_ans << "\t\t"
                  << (serial_ans - ans) / serial_ans << "\t\t\t" << speedup << "\n\n";
    }

    hipMalloc((void **)&IA_d, sizeof(long unsigned) * (n + 1));
    hipMalloc((void **)&JA_d, sizeof(long unsigned) * A.edge_count * 2);
    hipMalloc((void **)&spMV_ans_d, sizeof(T) * n);
    hipMemcpy(IA_d, A.row_offset, sizeof(long unsigned) * (n + 1), hipMemcpyHostToDevice);
    hipMemcpy(JA_d, A.col_idx, sizeof(long unsigned) * 2 * A.edge_count, hipMemcpyHostToDevice);
    {
        std::vector<T> gpu_ans_vec(n);

        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_spMV1<T><<<blocks, threads>>>(IA_d, JA_d, static_cast<unsigned long>(n), x_d, spMV_ans_d);

        hipMemcpy(&gpu_ans_vec[0], spMV_ans_d, sizeof(T) * n, hipMemcpyDeviceToHost);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);

        timeval start, end;
        gettimeofday(&start, NULL);
        spMV(A, &x[0], &ans_vec[0]);
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        T relative_error{0};
        unsigned max_idx{0u};
        diff_arrays(&gpu_ans_vec[0], &ans_vec[0], n, relative_error, max_idx);

        ans = std::sqrt(std::inner_product(gpu_ans_vec.begin(), gpu_ans_vec.end(), gpu_ans_vec.begin(), 0));
        auto serial_ans = std::sqrt(std::inner_product(ans_vec.begin(), ans_vec.end(), ans_vec.begin(), 0));

        std::cout << "SPMV1: \t\t" << ans
                  << "\t\t" << serial_ans << "\t\t"
                  << relative_error / serial_ans << "\t\t\t" << speedup << "\n\n";
    }
    {
        std::vector<T> gpu_ans_vec(n);
        std::vector<long unsigned> blockrows(n);
        long unsigned blocks_needed{0u};
        get_blockrows<T>(A, block_size, &blockrows[0], blocks_needed);

        dim3 blocks_IPCSR{static_cast<unsigned>(blocks_needed)};

        long unsigned *blockrows_d;

        hipMalloc((void **)&blockrows_d, sizeof(long unsigned) * (blocks_needed + 1));

        hipMemcpy(blockrows_d, &blockrows[0], sizeof(long unsigned) * (blocks_needed + 1), hipMemcpyHostToDevice);

        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_spMV2<T, long unsigned><<<blocks_IPCSR, threads>>>(IA_d, JA_d, blockrows_d, static_cast<unsigned long>(n), x_d, spMV_ans_d);

        hipMemcpy(&gpu_ans_vec[0], spMV_ans_d, sizeof(T) * n, hipMemcpyDeviceToHost);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);

        timeval start, end;
        gettimeofday(&start, NULL);
        spMV(A, &x[0], &ans_vec[0]);
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        T relative_error{0};
        unsigned max_idx{0u};
        diff_arrays(&gpu_ans_vec[0], &ans_vec[0], n, relative_error, max_idx);

        ans = std::sqrt(std::inner_product(gpu_ans_vec.begin(), gpu_ans_vec.end(), gpu_ans_vec.begin(), 0));
        auto serial_ans = std::sqrt(std::inner_product(ans_vec.begin(), ans_vec.end(), ans_vec.begin(), 0));

        std::cout << "SPMV2: \t\t" << ans
                  << "\t\t" << serial_ans << "\t\t"
                  << relative_error / serial_ans << "\t\t\t" << speedup << "\n\n";
        hipFree(blockrows_d);
    }
    {
        std::vector<T> gpu_ans_vec(n);
        std::vector<long unsigned> blockrows(n);

        T *tmp_d;

        hipMalloc((void **)&tmp_d, sizeof(T) * A.get_edges() * 2);

        hipEvent_t start_d, end_d;
        cuda_start_timer(start_d, end_d);

        cu_spMV3_kernel1<T, long unsigned><<<blocks, threads>>>(JA_d, A.get_edges() * 2, x_d, tmp_d);
        cu_spMV3_kernel2<T, long unsigned><<<1, threads, 49152>>>(tmp_d, IA_d, static_cast<unsigned long>(n), spMV_ans_d);

        hipMemcpy(&gpu_ans_vec[0], spMV_ans_d, sizeof(T) * n, hipMemcpyDeviceToHost);

        float gpu_time{cuda_end_timer(start_d, end_d)};

        hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);

        timeval start, end;
        gettimeofday(&start, NULL);
        spMV(A, &x[0], &ans_vec[0]);
        gettimeofday(&end, NULL);
        auto speedup{(end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0) / gpu_time};

        T relative_error{0};
        unsigned max_idx{0u};
        diff_arrays(&gpu_ans_vec[0], &ans_vec[0], n, relative_error, max_idx);

        ans = std::sqrt(std::inner_product(gpu_ans_vec.begin(), gpu_ans_vec.end(), gpu_ans_vec.begin(), 0));
        auto serial_ans = std::sqrt(std::inner_product(ans_vec.begin(), ans_vec.end(), ans_vec.begin(), 0));

        std::cout << "SPMV3: \t\t" << ans
                  << "\t\t" << serial_ans << "\t\t"
                  << relative_error / serial_ans << "\t\t\t" << speedup << "\n\n";

        hipFree(tmp_d);
    }

    hipFree(x_d);
    hipFree(y_d);
    hipFree(IA_d);
    hipFree(JA_d);
    hipFree(tmp_d);
    hipFree(ans_d);
    hipFree(spMV_ans_d);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    hipProfilerStop();
}

void cuda_start_timer(hipEvent_t &start, hipEvent_t &end)
{
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
}

float cuda_end_timer(hipEvent_t &start, hipEvent_t &end)
{
    hipEventRecord(end, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(end);
    float time_taken;
    hipEventElapsedTime(&time_taken, start, end);
    return time_taken * 0.001;
}