#include "hip/hip_runtime.h"
#include "../lib/cu_linalg.h"
#include "../lib/cu_SPMV.h"
#include "../lib/SPMV.h"
#include "../lib/adjMatrix.h"
#include "../lib/helpers.h"

#include <iostream>
#include <iomanip>
#include <random>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime_api.h>

#define BLOCKSIZE 32
#define SEED 1234       // To seed RNG
#define WIDTH 71        // for formatting std::cout output

template <typename T> void cu_linalg_test(const unsigned n);

int main (void)
{
    unsigned n{10000};
    
    std::cout << "\nTesting CUDA vs serial execution of linalg functions for n = "<<n<<"\n\n";
    
    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n' << std::setfill(' ');
    std::cout << "SINGLE PRECISION\n";
    cu_linalg_test<float>(n);
    
    std::cout << "\n";
    
    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n' << std::setfill(' ');
    std::cout << "DOUBLE PRECISION\n";
    cu_linalg_test<double>(n);

}


template <typename T>
void cu_linalg_test(const unsigned n) {
    T ans;
    
    std::random_device rd;
    std::mt19937 gen{SEED};
    //std::mt19937 gen{rd()};
    std::uniform_real_distribution<T> U(0.0, 1.0);

    std::vector<T> x(n), y(n), ans_vec(n), gpu_ans_vec(n);


    for (auto it = x.begin(); it != x.end(); it++)
        *it = U(gen);

    for (auto it = y.begin(); it != y.end(); it++)
        *it = U(gen);

    unsigned block_size{BLOCKSIZE}, num_blocks{n / block_size + 1};
    
    dim3 blocks{num_blocks}, threads{block_size}, one_block {1u};
    
    T *x_d, *y_d, *tmp_d, *ans_d;
    hipMalloc((void **)&x_d, sizeof(T) * n);
    hipMalloc((void **)&y_d, sizeof(T) * n);
    hipMalloc((void **)&tmp_d, sizeof(T) * num_blocks);
    hipMalloc((void **)&ans_d, sizeof(T));
    hipMemcpy(x_d, &x[0], sizeof(T) * n, hipMemcpyHostToDevice);
    hipMemcpy(y_d, &y[0], sizeof(T) * n, hipMemcpyHostToDevice);

    cu_dot_prod<T,BLOCKSIZE><<<blocks, threads, block_size*sizeof(T)>>>(x_d, y_d, n, tmp_d);
    cu_reduce<T,BLOCKSIZE><<<one_block, threads, num_blocks*sizeof(T)>>>(tmp_d, num_blocks, ans_d);

    hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);
    auto serial_ans = std::inner_product(x.begin(), x.end(), y.begin(), 0.0);

    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n' << std::setfill(' ');
    std::cout << "\t\t\tCUDA\t\tSerial\t\tRelative Error\n";
    std::cout << std::setw(WIDTH) << std::setfill('~') << '\n' << std::setfill(' ');

    std::cout << "Inner product: \t\t" << ans 
    << "\t\t" << serial_ans << "\t\t" 
    << (serial_ans - ans)/serial_ans << "\n\n";
    
    cu_norm_sq<T,BLOCKSIZE><<<blocks, threads, block_size*sizeof(T)>>>(x_d, n, tmp_d);
    cu_reduce<T,BLOCKSIZE><<<one_block, threads, num_blocks*sizeof(T)>>>(tmp_d, num_blocks, ans_d);

    hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);
    serial_ans = std::inner_product(x.begin(), x.end(), x.begin(), 0.0);
    
    std::cout << "Norm squared: \t\t" << ans 
    << "\t\t" << serial_ans << "\t\t" 
    << (serial_ans - ans)/serial_ans << "\n\n";

    cu_reduce<T,BLOCKSIZE><<<blocks, threads, block_size*sizeof(T)>>>(x_d, n, tmp_d);
    cu_reduce<T,BLOCKSIZE><<<one_block, threads, num_blocks*sizeof(T)>>>(tmp_d, num_blocks, ans_d);

    hipMemcpy(&ans, ans_d, sizeof(T), hipMemcpyDeviceToHost);

    serial_ans = std::accumulate(x.begin(), x.end(), 0.0);
    std::cout << "Reduce: \t\t" << ans 
    << "\t\t" << serial_ans << "\t\t" 
    << (serial_ans - ans)/serial_ans << "\n\n";

    long unsigned edges {n*10};
    adjMatrix A(n, edges);
    spMV(A, &x[0], &ans_vec[0]);

    long unsigned * IA_d, *JA_d;
    T * spMV_ans_d;
    hipMalloc((void**)&IA_d, sizeof(long unsigned)*(n+1));
    hipMalloc((void**)&JA_d, sizeof(long unsigned)*edges*2);
    hipMalloc((void**)&spMV_ans_d, sizeof(T)*n);
    hipMemcpy(IA_d, A.row_offset, sizeof(long unsigned)*(n+1),hipMemcpyHostToDevice);
    hipMemcpy(JA_d, A.col_idx, sizeof(long unsigned)*2*edges,hipMemcpyHostToDevice);
    
    long unsigned long_n {static_cast<long unsigned>(n)};
    
    cu_spMV1<T, long unsigned><<<blocks,threads>>>(IA_d, JA_d, long_n, x_d, spMV_ans_d);
    
    std::cout << "y[0]\t" << gpu_ans_vec[0] << '\n';
    hipMemcpy(&gpu_ans_vec[0], spMV_ans_d, sizeof(T)*n, hipMemcpyDeviceToHost);
    std::cout << "y[0]\t" << gpu_ans_vec[0] << '\n';

    T relative_error {0};
    unsigned max_idx {0u};
    diff_arrays(&gpu_ans_vec[0], &ans_vec[0], n, relative_error, max_idx);


    ans = std::sqrt(std::inner_product(gpu_ans_vec.begin(), gpu_ans_vec.end(), gpu_ans_vec.begin(), 0));
    serial_ans = std::sqrt(std::inner_product(ans_vec.begin(), ans_vec.end(), ans_vec.begin(), 0));
    
    std::cout << "SPMV: \t\t\t" << ans
    << "\t\t" << serial_ans << "\t\t" 
    << relative_error/serial_ans << "\n\n";

    hipFree(x_d);
    hipFree(y_d);
    hipFree(IA_d);
    hipFree(JA_d);
    hipFree(tmp_d);
    hipFree(ans_d);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    hipProfilerStop();
}