#include "hip/hip_runtime.h"
#include "multiplyOut.h"
#include "lapacke.h"
#include "cblas.h"
#include <iomanip>

void naive_dgemm(double * a, double * b, const unsigned rows, const unsigned cols, double * ans);
void naive_dgemv(double * a, double * v, const unsigned rows, const unsigned cols, double * ans);

void exp_func(double & a) {
        a = std::exp(a);
}

void multOut(lanczosDecomp & L, eigenDecomp & E, adjMatrix & A) {

        auto n {L.get_n()};
        
        // Applying function
        for (auto j=0u;j<L.krylov_dim;j++) exp_func(E.eigenvalues[j]);
        
        // Elementwise multiplying of f(lambda) by first row of eigenvectors
        for (auto j=0u;j<L.krylov_dim;j++) E.eigenvalues[j] *= L.x_norm * E.eigenvectors[j];

        //print_matrix(3, 1, &E.eigenvalues[0]);
        
        double * QV {new double [n*L.krylov_dim]};
        /*
        for (auto j = 0u; j < n; j++)
        {
                for (auto k = 0u; k < L.krylov_dim; k++)
                        std::cout << std::setprecision(20) << Q[k + j * L.krylov_dim] << " ";
                std::cout << '\n';
        }
        for (auto j = 0u; j < L.krylov_dim; j++)
        {
                for (auto k = 0u; k < L.krylov_dim; k++)
                        std::cout << std::setprecision(20) << E.eigenvectors[k + j * L.krylov_dim] << " ";
                std::cout << '\n';
        }
        */
/*
        // Getting QV (n x k)
        for (auto i=0u;i<n;i++) {
                for (auto j=0u;j<L.krylov_dim;j++) {
                        QV[i*L.krylov_dim+j] = 0.0;
                        for (auto k=0u;k<L.krylov_dim;k++) {
                                QV[i*L.krylov_dim+j] += L.Q[i*L.krylov_dim+k]*E.eigenvectors[k*L.krylov_dim+j];
                        }
                }
        }
        */
        // This call to cblas_dgemm was not working for me!
        auto k = L.get_krylov();
        //naive_dgemm(L.Q, E.eigenvectors, n,k, QV);
        cblas_dgemm (CblasRowMajor, CblasNoTrans, CblasNoTrans, n, k, k, 1, L.Q, k, E.eigenvectors, k, 0, QV, k);
// PRINT OUT QV
/*
        std::cout << "\nQV\n";
        for (auto j = 0u; j < L.krylov_dim; j++)
        {
                for (auto k = 0u; k < L.krylov_dim; k++)
                        std::cout <<std::setprecision(4)<< QV[k + j * L.krylov_dim] << " ";
                std::cout << '\n';
        }
        */
/*
        */
        // Getting QV*f(lambda)
        cblas_dgemv(CblasRowMajor, CblasNoTrans, n, L.krylov_dim, 1, QV, k, &E.eigenvalues[0], 1, 0, &L.ans[0],1);
        //naive_dgemv(QV, &E.eigenvalues[0], n, L.krylov_dim, &L.ans[0]);

        delete[](QV);
        
}

void print_matrix(unsigned rows, unsigned cols, double * A) {
        std::cout << "Printing matrix for "<<rows<<" rows and " << cols<< "cols\n";
        for (auto i=0u; i<rows; ++i) {
                for (auto j=0u; j<cols; ++j){
                        std::cout << A[i*cols+j] << " ";
                }
                std::cout << '\n';
        }
}

void naive_dgemm(double * a, double * b, const unsigned rows, const unsigned cols, double * ans) {
  for (int i=0;i<rows;i++) {
    for (int j=0;j<cols;j++) {
      ans[i*cols+j] = 0.0;
      for (int k=0;k<cols;k++) {
        ans[i*cols+j] += a[i*cols+k]*b[k*cols+j];
      }
    }
  }
}

void naive_dgemv(double * a, double * v, const unsigned rows, const unsigned cols, double * ans) {
  for (int i=0;i<rows;i++) {
    ans[i] = 0.0;
    for (int j=0;j<cols;j++) 
      ans[i] += a[i*cols+j]*v[j];
  }
}
