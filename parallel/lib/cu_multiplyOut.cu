#include "hip/hip_runtime.h"
#include "cu_multiplyOut.h"
#include "cblas.h"
#include <iomanip>
#include <algorithm>
#include <type_traits>

#include "hipblas.h"

  template <typename T>
void my_exp_func(T &a)
{
  a = std::exp(a);
}

  template <typename T>
void cu_multOut(lanczosDecomp<T> &L, eigenDecomp<T> &E, adjMatrix &A)
{

  auto n{L.get_n()}, k{L.get_krylov()};

  // Applying function
  for (auto j = 0u; j < L.krylov_dim; j++)
    my_exp_func(E.eigenvalues[j]);

  // Elementwise multiplying of f(lambda) by first row of eigenvectors
  for (auto j = 0u; j < L.krylov_dim; j++)
    E.eigenvalues[j] *= L.x_norm * E.eigenvectors[j];

  //print_matrix(3, 1, &E.eigenvalues[0]);
  T *Q_d, *V_d, *QV_d, *eigvals_d, *ans_d, alpha {1.0}, beta {0.0};

  std::vector<T> QV(n*k, 1.0);

  hipblasStatus_t status;
  hipError_t cudaStat;
  hipblasHandle_t handle;

  cudaStat = hipMalloc(&Q_d, sizeof(T) * n * k);
  if (cudaStat != hipSuccess) {
    std::cerr << "Allocation error for Q_d\n";
    return;
  }
  cudaStat = hipMalloc(&V_d, sizeof(T) * k * k);
  if (cudaStat != hipSuccess) {
    std::cerr << "Allocation error for V_d.\n";
    return;
  }
  cudaStat = hipMalloc(&QV_d, sizeof(T) * n * k);
  if (cudaStat != hipSuccess) {
    std::cerr << "Allocation error for QV_d.\n";
    return;
  }
  cudaStat = hipMalloc(&eigvals_d, sizeof(T) * k);
  if (cudaStat != hipSuccess) {
    std::cerr << "Allocation error for eigvals_d.\n";
    return;
  }
  cudaStat = hipMalloc(&ans_d, sizeof(T) * n);
  if (cudaStat != hipSuccess) {
    std::cerr << "Allocation error for ans_d.\n";
    return;
  }

  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Cublas initialization error.\n";
    return;
  }
  status = hipblasSetVector(n*k, sizeof(T),L.Q, 1, Q_d,1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Device access error.\n";
    return;
  }
  status = hipblasSetVector(k*k, sizeof(T),E.eigenvectors, 1, V_d,1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Device access error.\n";
    return;
  }
  status = hipblasSetVector(k, sizeof(T),E.eigenvalues, 1, eigvals_d,1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Device access error.\n";
    return;
  }
  status = hipblasSetVector(k*n, sizeof(T),&QV[0], 1, QV_d,1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Device access error.\n";
    return;
  }

  if (std::is_same<T, double>::value){
    // DGEMM
    status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,k,n,k,&alpha,V_d,k,Q_d,k,&beta,QV_d,k);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "Dgemm error.\n";
      return;
    }

    // DGEMV
    status = hipblasDgemv(handle, HIPBLAS_OP_T, k, n, &alpha, QV_d, k, eigvals_d, 1,&beta ,ans_d, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "Dgemv error.\n";
      return;
    }
  }
  if (std::is_same<T, float>::value){
    // DGEMM
    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,k,n,k,&alpha,V_d,k,Q_d,k,&beta,QV_d,k);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "Sgemm error.\n";
      return;
    }

    // DGEMV
    status = hipblasSgemv(handle, HIPBLAS_OP_T, k, n, &alpha, QV_d, k, eigvals_d, 1,&beta ,ans_d, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "Sgemv error.\n";
      return;
    }
  }

  status = hipblasGetVector(n, sizeof(T),ans_d, 1,&L.ans[0],1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Error transferring from device to host.\n";
    return;
  }

  hipblasDestroy(handle);

  hipFree(Q_d);
  hipFree(V_d);
  hipFree(QV_d);
  hipFree(eigvals_d);
  hipFree(ans_d);
}
