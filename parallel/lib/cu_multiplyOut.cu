#include "hip/hip_runtime.h"
#include "cu_multiplyOut.h"
#include "cblas.h"
#include <iomanip>
#include <algorithm>

#include "hipblas.h"

void my_exp_func(double &a)
{
        a = std::exp(a);
}

void cu_multOut(lanczosDecomp &L, eigenDecomp &E, adjMatrix &A)
{

        auto n{L.get_n()}, k{L.get_krylov()};

        // Applying function
        for (auto j = 0u; j < L.krylov_dim; j++)
                my_exp_func(E.eigenvalues[j]);

        // Elementwise multiplying of f(lambda) by first row of eigenvectors
        for (auto j = 0u; j < L.krylov_dim; j++)
                E.eigenvalues[j] *= L.x_norm * E.eigenvectors[j];

        //print_matrix(3, 1, &E.eigenvalues[0]);
        double *Q_d, *V_d, *QV_d, *eigvals_d, *ans_d, *alpha_d, *beta_d, alpha {1}, beta {0};

        hipMalloc(&Q_d, sizeof(double) * n * k);
        hipMalloc(&V_d, sizeof(double) * k * k);
        hipMalloc(&QV_d, sizeof(double) * n * k);
        hipMalloc(&eigvals_d, sizeof(double) * k);
        hipMalloc(&ans_d, sizeof(double) * n);
        hipMalloc(&alpha_d, sizeof(double));
        hipMalloc(&beta_d, sizeof(double));

        hipMemcpy(Q_d, L.Q, sizeof(double) * n * k, hipMemcpyHostToDevice);
        hipMemcpy(V_d, E.eigenvectors, sizeof(double) * k * k, hipMemcpyHostToDevice);
        hipMemcpy(eigvals_d, E.eigenvalues, sizeof(double) * k, hipMemcpyHostToDevice);
        hipMemcpy(alpha_d, &alpha, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(beta_d, &beta, sizeof(double), hipMemcpyHostToDevice);

        hipblasHandle_t handle{};


        hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,k,n,k,alpha_d,V_d,k,Q_d,n,beta_d,QV_d,n);
        hipblasDgemv(handle, HIPBLAS_OP_T, k, n, alpha_d, Q_d, k, eigvals_d, 1, beta_d ,ans_d, 1);

        std::vector<double> tmp(10);

        hipMemcpy(&tmp[0], V_d, sizeof(double) * 10, hipMemcpyDeviceToHost);
        std::cout << "CU first QV:\n";
        std::for_each(tmp.begin(), tmp.end(), [](double & a){ std::cout << a << " ";});
        std::cout << "\n";

        hipMemcpy(&L.ans[0], ans_d, sizeof(double) * n, hipMemcpyDeviceToHost);
        hipMemcpy(&tmp[0], ans_d, sizeof(double) * 10, hipMemcpyDeviceToHost);
        std::cout << "CU first ans:\n";
        std::for_each(tmp.begin(), tmp.end(), [](double & a){ std::cout << a << " ";});
        std::cout << "\n\n";

        hipFree(Q_d);
        hipFree(V_d);
        hipFree(QV_d);
        hipFree(eigvals_d);
        hipFree(ans_d);
        hipFree(alpha_d);
        hipFree(beta_d);
}
