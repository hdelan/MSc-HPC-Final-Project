#include "hip/hip_runtime.h"

/**
 * \file:        helpers.cu
 * \brief:       A few helper functions
 * \author:      Hugh Delaney
 * \version:     
 * \date:        2021-09-16
 */
#include "helpers.h"

#include <numeric>
#include <vector>
#include <cmath>

void cuda_start_timer(hipEvent_t &start, hipEvent_t &end)
{
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
}

float cuda_end_timer(hipEvent_t &start, hipEvent_t &end)
{
    hipEventRecord(end, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(end);
    float time_taken;
    hipEventElapsedTime(&time_taken, start, end);
    return time_taken * 0.001;
}

int parseArguments(int argc, char *argv[], std::string &filename, unsigned &krylov_dim, bool &verbose, unsigned &n, unsigned &bar_deg, unsigned &E)
{
        int c;

        while ((c = getopt(argc, argv, "k:f:b:n:e:v")) != -1)
        {
                switch (c)
                {
                case 'f':
                        filename = optarg;
                        break;
                case 'k':
                        krylov_dim = atoi(optarg);
                        break;
                case 'b':
                        bar_deg = atoi(optarg);
                        break;
                case 'n':
                        n = atoi(optarg);
                        break;
                case 'e':
                        E = atoi(optarg);
                        break;
                case 'v':
                        verbose = true;
                        break;
                default:
                        fprintf(stderr, "Invalid option given\n");
                        return -1;
                }
        }
        return 0;
}

template <typename T>
void diff_arrays(const T *const a, const T *const b, const unsigned n, T &relative_error, unsigned &max_entry)
{
        std::vector<T> diff(n);
        max_entry = 0u;
        for (auto i = 0u; i < n; i++)
        {
                diff[i] = a[i] - b[i];
                if (diff[i] > diff[max_entry])
                        max_entry = i;
        }
        relative_error = std::sqrt(std::inner_product(diff.begin(), diff.end(),diff.begin(),0))/norm(a, n);
}

template <typename T>
T norm(const T *const a, const unsigned n)
{
        T ans{0};
        for (auto i = 0u; i < n; i++)
        {
                ans += a[i] * a[i];
        }
        return std::sqrt(ans);
}
  template <typename T>
void my_exp_func(T &a)
{
  a = std::exp(a);
}
// Overloading cublas functions so I can call Sgemm through Dgemm 
hipblasStatus_t hipblasDgemm(hipblasHandle_t handle, hipblasOperation_t transa, hipblasOperation_t transb,
                          int m, int n, int k,
                          float * alpha,
                          float * A, int lda,
                          float * B, int ldb,
                          float * beta,
                          float * C, int ldc) {
  return hipblasSgemm(handle, transa, transb, m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
}
hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t transa,
                          int m, int n,
                          float * alpha,
                          float * A, int lda,
                          float * x, int incx,
                          float * beta,
                          float * y, int incy) {
  return hipblasSgemv(handle,transa,m,n,alpha,A,lda,x,incx,beta,y,incy);
}

void cblas_dgemm(CBLAS_ORDER layout, CBLAS_TRANSPOSE transa, CBLAS_TRANSPOSE transb, 
                        int m, int n, int k, 
                        float alpha, 
                        float * A, int lda, 
                        float * B, int ldb, 
                        float beta, 
                        float * C, int ldc) {
  cblas_sgemm (layout, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
void cblas_dgemv(CBLAS_ORDER layout, CBLAS_TRANSPOSE transa, 
                        int m, int n, 
                        float alpha,
                        float * A, int lda, 
                        float * x, int incx, 
                        float beta,
                        float * y, int incy) {
  cblas_sgemv(layout, transa, m, n, alpha, A, lda, x, incx, beta, y, incy);
}

template float norm<float>(const float *const a, const unsigned n);
template double norm<double>(const double *const a, const unsigned n);

template void diff_arrays<float>(const float *const, const float *const, const unsigned n, float &, unsigned &);
template void diff_arrays<double>(const double *const, const double *const, const unsigned n, double &, unsigned &);


template void my_exp_func(float &a);
template void my_exp_func(double &a);
