#include "hip/hip_runtime.h"
/**
 * \file:        cu_linalg.cu
 * \brief:       Linalg functions used in cu_lanczos.cu
 * \author:      Hugh Delaney
 * \version:     
 * \date:        2021-09-16
 */
#include "cu_linalg.h"
#include <stdio.h>

#define FULL_MASK 0xffffffff


/* --------------------------------------------------------------------------*/
/**
 * \brief:       Warp reduce function using shared memory
 *
 * @tparam T
 * @tparam blockSize
 * \param:       sdata
 * \param:       tid
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T, unsigned blockSize>
__device__ void warpReduce(volatile T * sdata, const unsigned tid) {
  if (blockSize >= 32) sdata[tid] += sdata[tid+16];
  if (blockSize >= 16) sdata[tid] += sdata[tid+8]; 
  if (blockSize >=  8) sdata[tid] += sdata[tid+4]; 
  if (blockSize >=  4) sdata[tid] += sdata[tid+2]; 
  if (blockSize >=  2) sdata[tid] += sdata[tid+1]; 
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       Warp reduce function using warp-level directives
 *
 * \param:       val
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T, unsigned blockSize>
__inline__ __device__ T warpReduceSum(T val) {
  if (blockSize >= 32) val += __shfl_down_sync(FULL_MASK, val, 16);
  if (blockSize >= 16) val += __shfl_down_sync(FULL_MASK, val,  8);
  if (blockSize >=  8) val += __shfl_down_sync(FULL_MASK, val,  4);
  if (blockSize >=  4) val += __shfl_down_sync(FULL_MASK, val,  2);
  if (blockSize >=  2) val += __shfl_down_sync(FULL_MASK, val,  1);
  return val;
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       Dot product
 *
 * \param:       a
 * \param:       b
 * \param:       n
 * \param:       ans
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T, unsigned blockSize>
__global__ void cu_dot_prod(T * a, T * b, const unsigned n, T * ans) {
  unsigned tid = threadIdx.x;
  int i = blockIdx.x*blockSize*2 + tid;
  unsigned gridSize = blockSize*2*gridDim.x;

  __shared__ T sdata[blockSize];
  sdata[tid]=0.0;

  while (i+blockSize<n) {
    sdata[tid] += a[i]*b[i] + a[i+blockSize]*b[i+blockSize];
    i += gridSize;
  }

  if (i<n) sdata[tid] += a[i]*b[i];

  __syncthreads();

  if (blockSize == 1024) {if (tid < 512) sdata[tid] += sdata[tid+512]; __syncthreads(); }
  if (blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid+256]; __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid+128]; __syncthreads(); }
  if (blockSize >= 128) { if (tid < 64) sdata[tid] += sdata[tid+64]; __syncthreads(); }
  if (blockSize >= 64)  { if (tid < 32) sdata[tid] += sdata[tid+32]; __syncthreads(); }

  if (tid < 32) sdata[tid] = warpReduceSum<T,blockSize>(sdata[tid]);
  if (tid == 0) ans[blockIdx.x] = sdata[0];
}


/* --------------------------------------------------------------------------*/
/**
 * \brief:       Reduce operation. Use for 1 block if used as second kernel in 
                 two part reduce.
 *
 * \param:       a
 * \param:       n
 * \param:       ans
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T, unsigned blockSize>
__global__ void cu_reduce(T * a, const unsigned n, T * ans) {
  unsigned tid = threadIdx.x;
  int i = blockIdx.x*blockSize*2 + tid;
  unsigned gridSize = blockSize*2*gridDim.x;

  __shared__ T sdata[blockSize];
  sdata[tid]=0.0;

  while (i+blockSize<n) {
    sdata[tid] += a[i]+a[i+blockSize];
    i += gridSize;
  }
  if (i<n) sdata[tid] += a[i];

  __syncthreads();

  if (blockSize == 1024) {if (tid < 512) sdata[tid] += sdata[tid+512]; __syncthreads(); }
  if (blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid+256]; __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid+128]; __syncthreads(); }
  if (blockSize >= 128) { if (tid < 64) sdata[tid] += sdata[tid+64]; __syncthreads(); }
  if (blockSize >= 64)  { if (tid < 32) sdata[tid] += sdata[tid+32]; __syncthreads(); }

  if (tid < 32) sdata[tid] = warpReduceSum<T,blockSize>(sdata[tid]);
  if (tid == 0) ans[blockIdx.x] = sdata[0];
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       A separate kernel for taking sqrt at the end. Useful as second
                 kernel for norm
 *
 * \param:       a
 * \param:       n
 * \param:       ans
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T, unsigned blockSize>
__global__ void cu_reduce_sqrt(T * a, const unsigned n, T * ans) {
  unsigned tid = threadIdx.x;
  int i = blockIdx.x*blockSize*2 + tid;
  unsigned gridSize = blockSize*2*gridDim.x;

  __shared__ T sdata[blockSize];
  sdata[tid]=0.0;

  while (i+blockSize<n) {
    sdata[tid] += a[i]+a[i+blockSize];
    i += gridSize;
  }
  if (i < n) sdata[tid] += a[i];

  __syncthreads();

  if (blockSize == 1024) {if (tid < 512) sdata[tid] += sdata[tid+512]; __syncthreads(); }
  if (blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid+256]; __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid+128]; __syncthreads(); }
  if (blockSize >= 128) { if (tid < 64) sdata[tid] += sdata[tid+64]; __syncthreads(); }
  if (blockSize >= 64)  { if (tid < 32) sdata[tid] += sdata[tid+32]; __syncthreads(); }

  if (tid < 32) sdata[tid] = warpReduceSum<T,blockSize>(sdata[tid]);
  if (tid == 0) ans[blockIdx.x] = std::sqrt(sdata[0]);
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       Norm squared
 *
 * \param:       a
 * \param:       n
 * \param:       ans
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T, unsigned blockSize>
__global__ void cu_norm_sq(T * a, const unsigned n, T * ans) {
  unsigned tid = threadIdx.x;
  int i = blockIdx.x*blockSize*2 + tid;
  unsigned gridSize = blockSize*2*gridDim.x;

  __shared__ T sdata[blockSize];
  sdata[tid]=0.0;

  while (i+blockSize<n) {
    sdata[tid] += a[i]*a[i]+a[i+blockSize]*a[i+blockSize];
    i += gridSize;
  }
  if (i<n) sdata[tid] += a[i]*a[i];

  __syncthreads();

  if (blockSize == 1024) {if (tid < 512) sdata[tid] += sdata[tid+512]; __syncthreads(); }
  if (blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid+256]; __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid+128]; __syncthreads(); }
  if (blockSize >= 128) { if (tid < 64) sdata[tid] += sdata[tid+64]; __syncthreads(); }
  if (blockSize >= 64)  { if (tid < 32) sdata[tid] += sdata[tid+32]; __syncthreads(); }

  if (tid < 32) sdata[tid] = warpReduceSum<T,blockSize>(sdata[tid]);
  if (tid == 0) ans[blockIdx.x] = sdata[0];
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       Norm squared sqrt
 *
 * \param:       a
 * \param:       n
 * \param:       ans
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T, unsigned blockSize>
__global__ void cu_norm_sq_sqrt(T * a, const unsigned n, T * ans) {
  unsigned tid = threadIdx.x;
  unsigned i = blockIdx.x*blockSize*2 + tid;
  unsigned gridSize = blockSize*2*gridDim.x;

  __shared__ T sdata[blockSize];

  sdata[tid]=0.0;

  while (i+blockSize<n) {
    sdata[tid] += a[i]*a[i]+a[i+blockSize]*a[i+blockSize];
    i += gridSize;
  }
  if (i<n) sdata[tid] += a[i]*a[i];

  __syncthreads();

  if (blockSize == 1024) {if (tid < 512) sdata[tid] += sdata[tid+512]; __syncthreads(); }
  if (blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid+256]; __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid+128]; __syncthreads(); }
  if (blockSize >= 128) { if (tid < 64) sdata[tid] += sdata[tid+64]; __syncthreads(); }
  if (blockSize >= 64)  { if (tid < 32) sdata[tid] += sdata[tid+32]; __syncthreads(); }

  if (tid < 32) sdata[tid] = warpReduceSum<T,blockSize>(sdata[tid]);
  if (tid == 0) ans[blockIdx.x] = std::sqrt(sdata[0]);
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       Computes v -= alpha*x_d
 *
 * \param:       v_d
 * \param:       alpha_d
 * \param:       x_d
 * \param:       n
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T>
__global__ void cu_dpax(T * v_d, T * alpha_d, T * x_d, const unsigned n) {
  unsigned tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid < n) v_d[tid] -= (*alpha_d)*x_d[tid];
}

/* --------------------------------------------------------------------------*/
/**
 * \brief:       Computes v = x/alpha
 *
 * \param:       v_d
 * \param:       alpha_d
 * \param:       x_d
 * \param:       n
 *
 * \returns      
 */
/* ----------------------------------------------------------------------------*/
template <typename T>
__global__ void cu_dvexda(T * v_d, T * alpha_d, T * x_d, const unsigned n) {
  unsigned tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid < n) v_d[tid] = x_d[tid]/(*alpha_d);
}

// EXPLICIT INSTANTIATIONS
template __global__ void cu_dpax<double>(double * v, double * alpha, double * x, const unsigned n);
template __global__ void cu_dpax<float>(float * v, float * alpha, float * x, const unsigned n);

template __global__ void cu_dvexda<double>(double * v, double * alpha, double * x, const unsigned n);
template __global__ void cu_dvexda<float>(float * v, float * alpha, float * x, const unsigned n);

template __device__ void warpReduce<float ,1>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 1>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 1>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 1>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 1>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 2>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 2>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 2>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 2>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 2>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 4>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 4>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 4>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 4>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 4>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 8>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 8>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 8>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 8>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 8>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 16>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 16>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 16>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 16>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 16>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 32>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 32>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 32>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 32>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 32>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 64>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 64>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 64>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 64>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 64>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 128>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 128>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 128>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 128>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 128>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 256>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 256>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 256>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 256>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 256>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 512>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 512>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 512>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 512>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 512>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<float, 1024>(volatile float * sdata, const unsigned tid);
template __global__ void cu_dot_prod<float, 1024>(float * a, float * b, const unsigned n, float * ans);
template __global__ void cu_reduce<float, 1024>(float * a, const unsigned n, float * ans);
template __global__ void cu_reduce_sqrt<float, 1024>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq<float, 1024>(float * a, const unsigned n, float * ans);

template __device__ void warpReduce<double, 1>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 1>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 1>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 1>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 1>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 2>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 2>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 2>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 2>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 2>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 4>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 4>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 4>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 4>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 4>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 8>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 8>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 8>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 8>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 8>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 16>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 16>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 16>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 16>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 16>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 32>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 32>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 32>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 32>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 32>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 64>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 64>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 64>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 64>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 64>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 128>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 128>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 128>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 128>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 128>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 256>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 256>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 256>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 256>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 256>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 512>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 512>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 512>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 512>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 512>(double * a, const unsigned n, double * ans);

template __device__ void warpReduce<double, 1024>(volatile double * sdata, const unsigned tid);
template __global__ void cu_dot_prod<double, 1024>(double * a, double * b, const unsigned n, double * ans);
template __global__ void cu_reduce<double, 1024>(double * a, const unsigned n, double * ans);
template __global__ void cu_reduce_sqrt<double, 1024>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq<double, 1024>(double * a, const unsigned n, double * ans);


template __global__ void cu_norm_sq_sqrt<float, 1>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 2>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 4>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 8>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 16>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 32>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 64>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 128>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 256>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 512>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<float, 1024>(float * a, const unsigned n, float * ans);
template __global__ void cu_norm_sq_sqrt<double, 1>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 2>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 4>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 8>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 16>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 32>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 64>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 128>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 256>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 512>(double * a, const unsigned n, double * ans);
template __global__ void cu_norm_sq_sqrt<double, 1024>(double * a, const unsigned n, double * ans);
