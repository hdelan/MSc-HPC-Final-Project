#include "hip/hip_runtime.h"
#include "cu_multiplyOut.h"
#include "multiplyOut.h"
#include "helpers.h"

#include <iomanip>
#include <algorithm>
#include <type_traits>

template <typename T>
__global__ void print_some(T * a) {
  printf("%E %E %E\n\n", a[0], a[1], a[2]);
}

template <typename T>
void cu_multOut(lanczosDecomp<T> &L, eigenDecomp<T> &E, adjMatrix &A)
{
  auto n{L.get_n()}, k{L.get_krylov()};

  // Applying function
  for (auto j = 0u; j < L.krylov_dim; j++)
    my_exp_func(E.eigenvalues[j]);

  // Elementwise multiplying of f(lambda) by first row of eigenvectors
  for (auto j = 0u; j < L.krylov_dim; j++)
    E.eigenvalues[j] *= L.x_norm * E.eigenvectors[j];

  //print_matrix(3, 1, &E.eigenvalues[0]);
  T *eigvals_d, *ans_d, alpha {1.0}, beta {0.0};

  cblas_dgemv(CblasRowMajor, CblasNoTrans, L.krylov_dim, L.krylov_dim, 1, &E.eigenvectors[0], k, &E.eigenvalues[0], 1, 0, &L.ans[0],1);

  hipblasStatus_t status;
  hipError_t cudaStat;
  hipblasHandle_t handle;

  hipStream_t stream[2];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);

  cudaStat = hipMalloc(&eigvals_d, sizeof(T) * k);
  if (cudaStat != hipSuccess) {
    std::cerr << "Allocation error for eigvals_d.\n";
    return;
  }
  cudaStat = hipMalloc(&ans_d, sizeof(T) * n);
  if (cudaStat != hipSuccess) {
    std::cerr << "Allocation error for ans_d.\n";
    return;
  }

  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Cublas initialization error.\n";
    return;
  }
  
  // Memory transfers
  status = hipblasSetVectorAsync(k, sizeof(T),&L.ans[0], 1, eigvals_d,1, stream[0]);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Device access error.\n";
    return;
  }
  //printf("\nSome values from Q_d: ");
  //print_some<<<1,1>>>(L.Q_d);
  /*
    // DGEMM
    status = hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
                                n,k,k,
                                &alpha,
                                L.Q_d,n,
                                V_d,k,
                                &beta,
                                QV_d,n);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "Dgemm error.\n";
      return;
    }
    //printf("\nSome values from QV_d: ");
    //print_some<<<1,1>>>(QV_d);
*/
    // DGEMV
    status = hipblasDgemv(handle,HIPBLAS_OP_N,
                                n,k,
                                &alpha,
                                L.Q_d,n,
                                eigvals_d,1,
                                &beta,ans_d,1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "Dgemv error.\n";
      return;
    }
    //printf("\nSome values from ans_d: ");
    //print_some<<<1,1>>>(ans_d);

  status = hipblasGetVector(n, sizeof(T),ans_d, 1,&L.ans[0],1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Error transferring from device to host.\n";
    return;
  }

  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  hipblasDestroy(handle);

  hipFree(eigvals_d);
  hipFree(ans_d);
}

template void cu_multOut(lanczosDecomp<float> &L, eigenDecomp<float> &E, adjMatrix &A);
template void cu_multOut(lanczosDecomp<double> &L, eigenDecomp<double> &E, adjMatrix &A);
