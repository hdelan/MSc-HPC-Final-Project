#include "hip/hip_runtime.h"
#include "cu_lanczos.h"

#define SPMV_BLOCKSIZE 1024
#define DOT_BLOCKSIZE 128
#define NORM_BLOCKSIZE 128
#define RED_BLOCKSIZE 256
#define SAX_BLOCKSIZE 256


  template <typename T>
void lanczosDecomp<T>::cu_decompose()
{
  unsigned n{A.get_n()};
  unsigned *IA_d, *JA_d, *blockrows_d;
  T *v_d, *alpha_d, *beta_d, *tmp_d;

  unsigned dot_grid {n/(DOT_BLOCKSIZE*2) + (n%(DOT_BLOCKSIZE*2)==0?0:1)};
  unsigned norm_grid {n/(NORM_BLOCKSIZE*2) + (n%(NORM_BLOCKSIZE*2)==0?0:1)};
  unsigned sax_grid {n/SAX_BLOCKSIZE + (n%SAX_BLOCKSIZE==0?0:1)};

  std::vector<unsigned> blockrows(n);
  unsigned spmv_blocks_needed {0u};

  get_blockrows<T>(A,SPMV_BLOCKSIZE,&blockrows[0],spmv_blocks_needed);

  T *x_normed{new T[n]};
  T x_norm = norm(x, n);

  for (auto k = 0u; k < n; k++)
    x_normed[k] = x[k] / x_norm;
  
  hipMalloc((void **)&Q_d, sizeof(T) * n * krylov_dim);

  hipMalloc((void **)&IA_d, sizeof(unsigned) * (n + 1));
  hipMalloc((void **)&JA_d, sizeof(unsigned) * 2 * A.edge_count);
  hipMalloc((void **)&v_d, sizeof(T) * n);
  hipMalloc((void **)&alpha_d, sizeof(T) * krylov_dim);
  hipMalloc((void **)&beta_d, sizeof(T) * (krylov_dim - 1));
  hipMalloc((void **)&tmp_d, sizeof(T) * (norm_grid));
  hipMalloc((void **)&blockrows_d, sizeof(unsigned) * (spmv_blocks_needed+1));

  auto global_memory_used { (n+1+2*A.edge_count)*sizeof(unsigned)+(n+2*krylov_dim-1+norm_grid)*sizeof(T)};

  std::cout << "\nUsing " 
        << global_memory_used << " bytes of CUDA global memory (" 
        << 100* global_memory_used/(double)11996954624 
        << "% of capacity 11996954624 bytes)\n";

  T *Q_d_ptr[krylov_dim];
  for (auto i=0u;i<krylov_dim;i++) Q_d_ptr[i] = &Q_d[n*i];

  std::vector<T> tmp(n);
  
  hipStream_t stream[3];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);

  hipMemcpyAsync(Q_d_ptr[0], x_normed, sizeof(T) * n, hipMemcpyHostToDevice, stream[0]);
  hipMemcpyAsync(IA_d, A.row_offset, sizeof(unsigned) * (n + 1), hipMemcpyHostToDevice, stream[1]);
  hipMemcpyAsync(JA_d, A.col_idx, sizeof(unsigned) * 2 * A.edge_count, hipMemcpyHostToDevice, stream[2]);
  hipMemcpyAsync(blockrows_d, &blockrows[0], sizeof(unsigned)*(spmv_blocks_needed+1), hipMemcpyHostToDevice, stream[0]);

  hipStreamSynchronize(stream[0]);
  hipStreamSynchronize(stream[1]);
  hipStreamSynchronize(stream[2]);

  hipStreamDestroy(stream[2]);

  for (auto k = 0u; k < krylov_dim; k++)
  {
    // v = A*Q(:,j)
    cu_spMV2<T, unsigned, SPMV_BLOCKSIZE><<<spmv_blocks_needed, SPMV_BLOCKSIZE>>>(IA_d, JA_d, blockrows_d, n, Q_d_ptr[k], v_d);
    //cu_spMV1<T><<<n/256+1,256>>>(IA_d, JA_d, n, Q_d_ptr[k], v_d);

    // alpha = v*Q(:,j)
    cu_dot_prod<T, DOT_BLOCKSIZE><<<dot_grid, DOT_BLOCKSIZE,0,stream[0]>>>(v_d, Q_d_ptr[k], n, tmp_d);
    cu_reduce<T, RED_BLOCKSIZE><<<1, RED_BLOCKSIZE, 0, stream[0]>>>(tmp_d, dot_grid, &alpha_d[k]);

    // v = v - alpha*Q(:,j)
    cu_dpax<T><<<sax_grid, SAX_BLOCKSIZE,0,stream[0]>>>(v_d, &alpha_d[k], Q_d_ptr[k], n);

    if (k > 0)
    {
      // v = v - beta*Q(:,j-1)
      cu_dpax<T><<<sax_grid, SAX_BLOCKSIZE,0,stream[0]>>>(v_d, &beta_d[k - 1], Q_d_ptr[k-1], n);
    }

    if (k < krylov_dim - 1)
    {
      // beta[j] = norm(v)
      cu_norm_sq<T, NORM_BLOCKSIZE><<<norm_grid, NORM_BLOCKSIZE, 0, stream[0]>>>(v_d, n, tmp_d);
      cu_reduce_sqrt<T,RED_BLOCKSIZE><<<1, RED_BLOCKSIZE, 0, stream[0]>>>(tmp_d, norm_grid, &beta_d[k]);
      
      // Q(:,j) = v/beta
      cu_dvexda<T><<<sax_grid,SAX_BLOCKSIZE,0,stream[0]>>>(Q_d_ptr[k+1], &beta_d[k], v_d, n);
    }
  }
  hipMemcpyAsync(alpha, alpha_d, sizeof(T) * krylov_dim, hipMemcpyDeviceToHost, stream[0]);
  hipMemcpyAsync(beta, beta_d, sizeof(T) * (krylov_dim - 1), hipMemcpyDeviceToHost, stream[1]);

  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);
  
  hipFree(IA_d);
  hipFree(JA_d);
  hipFree(v_d);
  hipFree(alpha_d);
  hipFree(beta_d);
  hipFree(tmp_d);
  hipFree(blockrows_d);
}

template void lanczosDecomp<float>::cu_decompose();
template void lanczosDecomp<double>::cu_decompose();

